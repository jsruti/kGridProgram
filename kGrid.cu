#include "hip/hip_runtime.h"
/*
* Compile: nvcc kGrid.cu -o kGrid.out
* Run: ./kGrid.out
*/
#include <stdio.h>
__global__
void kGrid(int n, int *k) {
	int l = blockIdx.x * blockDim.x + threadIdx.x;
	if(l < n) {
		for(int* i = 0; i < k; i++) {
			for(int* j = 0; j < k; j++) {
				printf("%i", j)
			}
		printf("\n");
		} 
	}
}

int main(void) {
	int N = 1<<20;
	int *k, *d_k;
	k = (int*)malloc(N*sizeof(int));
	
	hipMalloc(&d_k, N*sizeof(int));

	for(int i = 0; i < N; i++) {
		k[i] = 5;
	}
	
	hipMemcpy(d_k, k, N*sizeof(int), hipMemcpyHostToDevice);
	
	kGrid<<<(N+255)/256, 256>>>(N, d_k);
	
	hipMemcpy(k, d_k, N*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_k);
	free(k);

	return 0;
}

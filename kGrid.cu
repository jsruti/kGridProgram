/*
* Compile and run: nvcc -arch=sm_20 kGrid.cu -run
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
/*__global__
void kGrid(int n, int *k) {
	int l = blockIdx.x * blockDim.x + threadIdx.x;
	if(l < n) {
		for(int* i = 0; i < k; i++) {
			for(int* j = 0; j < k; j++) {
				printf("%i", j)
			}
		printf("\n");
		} 
	}
}

int main(void) {
	int N = 1<<20;
	int *k, *d_k;
	k = (int*)malloc(N*sizeof(int));
	
	cudaMalloc(&d_k, N*sizeof(int));

	for(int i = 0; i < N; i++) {
		k[i] = 5;
	}
	
	cudaMemcpy(d_k, k, N*sizeof(int), cudaMemcpyHostToDevice);
	
	kGrid<<<(N+255)/256, 256>>>(N, d_k);
	
	cudaMemcpy(k, d_k, N*sizeof(int), cudaMemcpyDeviceToHost);

	cudaFree(d_k);
	free(k);

	return 0;
}*/
__global__ void print_kernel() {
    printf("Block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    print_kernel<<<10, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
